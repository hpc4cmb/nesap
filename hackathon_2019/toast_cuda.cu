#include "hip/hip_runtime.h"

#include <utils.hpp>
#include <pointing_cuda.hpp>
#include <solver_cuda.hpp>

#include <cmath>
#include <sstream>
#include <iostream>
#include <cstring>

#include <hip/hip_runtime.h>

#ifdef HAVE_NVTX
#include <nvToolsExt.h>
#endif

// 2/PI
#define TWOINVPI 0.63661977236758134308

// 2/3
#define TWOTHIRDS 0.66666666666666666667


static void CudaError(hipError_t err, char const * file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
               file, line);
        exit(EXIT_FAILURE);
    }
}
#define CUDA_CHECK(err) (CudaError(err, __FILE__, __LINE__))

// Healpix operations needed for this test.

typedef struct {
    int64_t nside;
    int64_t npix;
    int64_t ncap;
    double dnside;
    int64_t twonside;
    int64_t fournside;
    int64_t nsideplusone;
    int64_t nsideminusone;
    double halfnside;
    double tqnside;
    int64_t factor;
    int64_t jr[12];
    int64_t jp[12];
    uint64_t utab[0x100];
    uint64_t ctab[0x100];
} hpix;

void hpix_init(hpix * hp, int64_t nside) {
    hp->nside = nside;
    hp->ncap = 2 * (nside * nside - nside);
    hp->npix = 12 * nside * nside;
    hp->dnside = static_cast <double> (nside);
    hp->twonside = 2 * nside;
    hp->fournside = 4 * nside;
    hp->nsideplusone = nside + 1;
    hp->halfnside = 0.5 * (hp->dnside);
    hp->tqnside = 0.75 * (hp->dnside);
    hp->factor = 0;
    hp->nsideminusone = nside - 1;
    while (nside != (1ll << hp->factor)) {
        ++hp->factor;
    }

    static const int64_t init_jr[12] = {2, 2, 2, 2, 3, 3, 3, 3, 4, 4, 4, 4};
    memcpy(hp->jr, init_jr, sizeof(init_jr));

    static const int64_t init_jp[12] = {1, 3, 5, 7, 0, 2, 4, 6, 1, 3, 5, 7};
    memcpy(hp->jp, init_jp, sizeof(init_jp));

    for (uint64_t m = 0; m < 0x100; ++m) {
        hp->utab[m] = (m & 0x1) | ((m & 0x2) << 1) | ((m & 0x4) << 2) |
                   ((m & 0x8) << 3) | ((m & 0x10) << 4) | ((m & 0x20) << 5) |
                   ((m & 0x40) << 6) | ((m & 0x80) << 7);

        hp->ctab[m] = (m & 0x1) | ((m & 0x2) << 7) | ((m & 0x4) >> 1) |
                   ((m & 0x8) << 6) | ((m & 0x10) >> 2) | ((m & 0x20) << 5) |
                   ((m & 0x40) >> 3) | ((m & 0x80) << 4);
    }
    return;
}

__device__ uint64_t hpix_xy2pix(hpix * hp, uint64_t x, uint64_t y) {
    return hp->utab[x & 0xff] | (hp->utab[(x >> 8) & 0xff] << 16) |
           (hp->utab[(x >> 16) & 0xff] << 32) |
           (hp->utab[(x >> 24) & 0xff] << 48) |
           (hp->utab[y & 0xff] << 1) | (hp->utab[(y >> 8) & 0xff] << 17) |
           (hp->utab[(y >> 16) & 0xff] << 33) |
           (hp->utab[(y >> 24) & 0xff] << 49);
}

__device__ void hpix_vec2zphi(hpix * hp, double const * vec,
                              double * phi, int * region, double * z,
                              double * rtz) {
    // region encodes BOTH the sign of Z and whether its
    // absolute value is greater than 2/3.
    (*z) = vec[2];
    double za = fabs(*z);
    int itemp = ((*z) > 0.0) ? 1 : -1;
    (*region) = (za <= TWOTHIRDS) ? itemp : itemp + itemp;
    (*rtz) = sqrt(3.0 * (1.0 - za));
    (*phi) = atan2(vec[1], vec[0]);
    return;
}

__device__ void hpix_zphi2nest(hpix * hp, double phi, int region, double z,
                               double rtz, int64_t * pix) {
    double tt = (phi >= 0.0) ? phi * TWOINVPI : phi * TWOINVPI + 4.0;
    int64_t x;
    int64_t y;
    double temp1;
    double temp2;
    int64_t jp;
    int64_t jm;
    int64_t ifp;
    int64_t ifm;
    int64_t face;
    int64_t ntt;
    double tp;

    if ((region == 1) || (region == -1)) {
        temp1 = hp->halfnside + hp->dnside * tt;
        temp2 = hp->tqnside * z;

        jp = (int64_t)(temp1 - temp2);
        jm = (int64_t)(temp1 + temp2);

        ifp = jp >> hp->factor;
        ifm = jm >> hp->factor;

        if (ifp == ifm) {
            face = (ifp == 4) ? (int64_t)4 : ifp + 4;
        } else if (ifp < ifm) {
            face = ifp;
        } else {
            face = ifm + 8;
        }

        x = jm & hp->nsideminusone;
        y = hp->nsideminusone - (jp & hp->nsideminusone);
    } else {
        ntt = (int64_t)tt;

        tp = tt - (double)ntt;

        temp1 = hp->dnside * rtz;

        jp = (int64_t)(tp * temp1);
        jm = (int64_t)((1.0 - tp) * temp1);

        if (jp >= hp->nside) {
            jp = hp->nsideminusone;
        }
        if (jm >= hp->nside) {
            jm = hp->nsideminusone;
        }

        if (z >= 0) {
            face = ntt;
            x = hp->nsideminusone - jm;
            y = hp->nsideminusone - jp;
        } else {
            face = ntt + 8;
            x = jp;
            y = jm;
        }
    }

    uint64_t sipf = hpix_xy2pix(hp, (uint64_t)x, (uint64_t)y);

    (*pix) = (int64_t)sipf + (face << (2 * hp->factor));

    return;
}

__device__ void hpix_zphi2ring(hpix * hp, double phi, int region, double z,
                               double rtz, int64_t * pix) {
    double tt = (phi >= 0.0) ? phi * TWOINVPI : phi * TWOINVPI + 4.0;
    double tp;
    int64_t longpart;
    double temp1;
    double temp2;
    int64_t jp;
    int64_t jm;
    int64_t ip;
    int64_t ir;
    int64_t kshift;

    if ((region == 1) || (region == -1)) {
        temp1 = hp->halfnside + hp->dnside * tt;
        temp2 = hp->tqnside * z;

        jp = (int64_t)(temp1 - temp2);
        jm = (int64_t)(temp1 + temp2);

        ir = hp->nsideplusone + jp - jm;
        kshift = 1 - (ir & 1);

        ip = (jp + jm - hp->nside + kshift + 1) >> 1;
        ip = ip % hp->fournside;

        (*pix) = hp->ncap + ((ir - 1) * hp->fournside + ip);
    } else {
        tp = tt - floor(tt);

        temp1 = hp->dnside * rtz;

        jp = (int64_t)(tp * temp1);
        jm = (int64_t)((1.0 - tp) * temp1);
        ir = jp + jm + 1;
        ip = (int64_t)(tt * (double)ir);
        longpart = (int64_t)(ip / (4 * ir));
        ip -= longpart;

        (*pix) = (region > 0) ? (2 * ir * (ir - 1) + ip)
                 : (hp->npix - 2 * ir * (ir + 1) + ip);
    }

    return;
}

__device__ void hpix_vec2nest(hpix * hp, double const * vec, int64_t * pix) {
    double z;
    double rtz;
    double phi;
    int region;
    hpix_vec2zphi(hp, vec, &phi, &region, &z, &rtz);
    hpix_zphi2nest(hp, phi, region, z, rtz, pix);
    return;
}

__device__ void hpix_vec2ring(hpix * hp, double const * vec, int64_t * pix) {
    double z;
    double rtz;
    double phi;
    int region;
    hpix_vec2zphi(hp, vec, &phi, &region, &z, &rtz);
    hpix_zphi2ring(hp, phi, region, z, rtz, pix);
    return;
}


// Quaternion operations needed for this test

__host__ void qa_normalize_inplace(size_t n, double * q) {
    for (size_t i = 0; i < n; ++i) {
        size_t off = 4 * i;
        double norm = 0.0;
        for (size_t j = 0; j < 4; ++j) {
            norm += q[off + j] * q[off + j];
        }
        norm = 1.0 / ::sqrt(norm);
        for (size_t j = 0; j < 4; ++j) {
            q[off + j] *= norm;
        }
    }
    return;
}

__device__ void qa_rotate(double const * q_in, double const * v_in,
                          double * v_out) {
    // The input quaternion has already been normalized on the host.

    double xw =  q_in[3] * q_in[0];
    double yw =  q_in[3] * q_in[1];
    double zw =  q_in[3] * q_in[2];
    double x2 = -q_in[0] * q_in[0];
    double xy =  q_in[0] * q_in[1];
    double xz =  q_in[0] * q_in[2];
    double y2 = -q_in[1] * q_in[1];
    double yz =  q_in[1] * q_in[2];
    double z2 = -q_in[2] * q_in[2];

    v_out[0] = 2 * ((y2 + z2) * v_in[0] + (xy - zw) * v_in[1] +
                            (yw + xz) * v_in[2]) + v_in[0];

    v_out[1] = 2 * ((zw + xy) * v_in[0] + (x2 + z2) * v_in[1] +
                            (yz - xw) * v_in[2]) + v_in[1];

    v_out[2] = 2 * ((xz - yw) * v_in[0] + (xw + yz) * v_in[1] +
                            (x2 + y2) * v_in[2]) + v_in[2];

    return;
}

__device__ void qa_mult(double const * p, double const * q, double * r) {
    r[0] =  p[0] * q[3] + p[1] * q[2] -
               p[2] * q[1] + p[3] * q[0];
    r[1] = -p[0] * q[2] + p[1] * q[3] +
               p[2] * q[0] + p[3] * q[1];
    r[2] =  p[0] * q[1] - p[1] * q[0] +
               p[2] * q[3] + p[3] * q[2];
    r[3] = -p[0] * q[0] - p[1] * q[1] -
               p[2] * q[2] + p[3] * q[3];
    return;
}


__device__ void stokes_weights(double hwpang, double cal, double eta,
                               double const * dir, double const * orient,
                               float * weights) {
    double by = orient[0] * dir[1] - orient[1] * dir[0];
    double bx = orient[0] * (-dir[2] * dir[0]) +
         orient[1] * (-dir[2] * dir[1]) +
         orient[2] * (dir[0] * dir[0] + dir[1] * dir[1]);
    double ang = atan2(by, bx);
    ang += 2.0 * hwpang;
    ang *= 2.0;
    double sang;
    double cang;
    sincos(ang, &sang, &cang);

    weights[0] = __double2float_rn(cal);
    weights[1] = __double2float_rn(cang * eta * cal);
    weights[2] = __double2float_rn(sang * eta * cal);
    return;
}


__global__ void single_detector_nest(
        hpix * hp,
        double cal,
        double eps,
        double const * detquat,
        int nsamp,
        double const * hwpang,
        double const * boresight,
        int64_t * detpixels,
        float * detweights
    ) {
    // This is the kernel function that works on one detector for some
    // number of samples.
    double xaxis[3] = {1.0, 0.0, 0.0};
    double zaxis[3] = {0.0, 0.0, 1.0};
    double eta = (1.0 - eps) / (1.0 + eps);
    double dir[3];
    double orient[3];
    double quat[4];

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nsamp;
         i += blockDim.x * gridDim.x) {
        qa_mult(&(boresight[4 * i]), detquat, quat);
        qa_rotate(quat, zaxis, dir);
        hpix_vec2nest(hp, dir, &(detpixels[i]));
        qa_rotate(quat, xaxis, orient);
        stokes_weights(hwpang[i], cal, eta, dir, orient, &(detweights[3 * i]));
    }
    return;
}


__global__ void single_detector_ring(
        hpix * hp,
        double cal,
        double eps,
        double const * detquat,
        int nsamp,
        double const * hwpang,
        double const * boresight,
        int64_t * detpixels,
        float * detweights
    ) {
    // This is the kernel function that works on one detector for some
    // number of samples.
    double xaxis[3] = {1.0, 0.0, 0.0};
    double zaxis[3] = {0.0, 0.0, 1.0};
    double eta = (1.0 - eps) / (1.0 + eps);
    double dir[3];
    double orient[3];
    double quat[4];

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nsamp;
         i += blockDim.x * gridDim.x) {
        qa_mult(&(boresight[4 * i]), detquat, quat);
        qa_rotate(quat, zaxis, dir);
        hpix_vec2ring(hp, dir, &(detpixels[i]));
        qa_rotate(quat, xaxis, orient);
        stokes_weights(hwpang[i], cal, eta, dir, orient, &(detweights[3 * i]));
    }
    return;
}


__host__ void detector_pointing_healpix(
        int64_t nside, bool nest,
        toast::AlignedVector <double> const & boresight,
        toast::AlignedVector <double> const & hwpang,
        toast::AlignedVector <std::string> const & detnames,
        std::map <std::string, toast::AlignedVector <double> > const & detquat,
        std::map <std::string, double> const & detcal,
        std::map <std::string, double> const & deteps,
        int numSMs, hipStream_t * streams, hpix * dev_hp,
        double * host_boresight, double * host_hwpang, double * host_detquat,
        double * dev_boresight, double * dev_hwpang, double * dev_detquat,
        int64_t * host_detpixels, float * host_detweights,
        int64_t * dev_detpixels, float * dev_detweights,
        std::map <std::string, toast::AlignedVector <int64_t> > & detpixels,
        std::map <std::string, toast::AlignedVector <double> > & detweights) {

    size_t nsamp = (size_t)(boresight.size() / 4);
    if (hwpang.size() != nsamp) {
        std::ostringstream o;
        o << "hwpang size not consistent with boresight.";
        throw std::runtime_error(o.str().c_str());
    }

    size_t ndet = detnames.size();
    if (detquat.size() != ndet) {
        std::ostringstream o;
        o << "number of det quaternions not consistent with number of names.";
        throw std::runtime_error(o.str().c_str());
    }
    if (detcal.size() != ndet) {
        std::ostringstream o;
        o << "number of det cal values not consistent with number of names.";
        throw std::runtime_error(o.str().c_str());
    }
    if (deteps.size() != ndet) {
        std::ostringstream o;
        o << "number of det eps vals not consistent with number of names.";
        throw std::runtime_error(o.str().c_str());
    }

    for (size_t d = 0; d < ndet; ++d) {
        if (detquat.count(detnames[d]) == 0) {
            std::ostringstream o;
            o << "no quaternion for det " << detnames[d];
            throw std::runtime_error(o.str().c_str());
        }
        if (detcal.count(detnames[d]) == 0) {
            std::ostringstream o;
            o << "no cal value for det " << detnames[d];
            throw std::runtime_error(o.str().c_str());
        }
        if (deteps.count(detnames[d]) == 0) {
            std::ostringstream o;
            o << "no epsilon value for det " << detnames[d];
            throw std::runtime_error(o.str().c_str());
        }
        if (detpixels.count(detnames[d]) == 0) {
            std::ostringstream o;
            o << "no pixel vector for det " << detnames[d];
            throw std::runtime_error(o.str().c_str());
        }
        if (detpixels.at(detnames[d]).size() != nsamp) {
            std::ostringstream o;
            o << "wrong size pixel vector for det " << detnames[d];
            throw std::runtime_error(o.str().c_str());
        }
        if (detweights.count(detnames[d]) == 0) {
            std::ostringstream o;
            o << "no weight vector for det " << detnames[d];
            throw std::runtime_error(o.str().c_str());
        }
        if (detweights.at(detnames[d]).size() != (3*nsamp)) {
            std::ostringstream o;
            o << "wrong size weight vector for det " << detnames[d];
            throw std::runtime_error(o.str().c_str());
        }
    }

    hipEvent_t sevents[ndet];
    for (size_t d = 0; d < ndet; ++d) {
        CUDA_CHECK(
            hipEventCreateWithFlags(&(sevents[d]), hipEventDisableTiming));
    }


    for (size_t d = 0; d < ndet; ++d) {
        std::memcpy(&(host_detquat[d * 4]), detquat.at(detnames[d]).data(),
                    4 * sizeof(double));
    }

    std::memcpy(host_boresight, boresight.data(), 4 * nsamp * sizeof(double));

    std::memcpy(host_hwpang, hwpang.data(), nsamp * sizeof(double));

    qa_normalize_inplace(nsamp, host_boresight);

    CUDA_CHECK(hipMemcpy(dev_boresight, host_boresight,
                          4 * nsamp * sizeof(double),
                          hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemcpy(dev_hwpang, host_hwpang, nsamp * sizeof(double),
               hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemcpy(dev_detquat, host_detquat, ndet * 4 * sizeof(double),
               hipMemcpyHostToDevice));

    // As a starting point, assume we run on the whole timestream for each
    // detector.  This may begin to approach the memory limits on the GPU,
    // at which point it is easy to add an outer loop here over chunks of
    // samples.

    // Threads per block
    int tpb = 256;

    // Blocks per Grid
    // int bpg = (int)((nsamp + tpb - 1) / tpb);
    int bpg = 32 * numSMs;

    for (size_t d = 0; d < ndet; ++d) {
        if (nest) {
            single_detector_nest <<<bpg, tpb, 0, streams[d]>>> (
                dev_hp,
                detcal.at(detnames[d]),
                deteps.at(detnames[d]),
                &(dev_detquat[4 * d]),
                nsamp,
                dev_hwpang,
                dev_boresight,
                &(dev_detpixels[d * nsamp]),
                &(dev_detweights[d * 3 * nsamp])
            );
        } else {
            single_detector_ring <<<bpg, tpb, 0, streams[d]>>> (
                dev_hp,
                detcal.at(detnames[d]),
                deteps.at(detnames[d]),
                &(dev_detquat[4 * d]),
                nsamp,
                dev_hwpang,
                dev_boresight,
                &(dev_detpixels[d * nsamp]),
                &(dev_detweights[d * 3 * nsamp])
            );
        }

        // memcopy results to host data structure.
        CUDA_CHECK(
            hipMemcpyAsync(&(host_detpixels[d * nsamp]),
                            &(dev_detpixels[d * nsamp]),
                            nsamp * sizeof(int64_t),
                            hipMemcpyDeviceToHost, streams[d]));
        CUDA_CHECK(
            hipMemcpyAsync(&(host_detweights[d * 3 * nsamp]),
                            &(dev_detweights[d * 3 * nsamp]),
                            3 * nsamp * sizeof(float),
                            hipMemcpyDeviceToHost, streams[d]));

        // Set event here so we can check when a stream is complete.
        CUDA_CHECK(hipEventRecord(sevents[d], streams[d]));
    }

    // Loop over streams and process completed ones until they are all done.
    size_t nfinished = 0;
    std::vector <bool> is_done(ndet);
    for (size_t d = 0; d < ndet; ++d) {
        is_done[d] = false;
    }
    while (nfinished != ndet) {
        for (size_t d = 0; d < ndet; ++d) {
            if (! is_done[d]) {
                if (hipEventQuery(sevents[d]) == hipSuccess) {
                    #ifdef HAVE_NVTX
                    nvtxRangePushA("memcpy");
                    #endif
                    std::memcpy(detpixels[detnames[d]].data(),
                                &(host_detpixels[d * nsamp]),
                                nsamp * sizeof(int64_t));
                    auto & dw = detweights[detnames[d]];
                    size_t woff = d * 3 * nsamp;
                    for (size_t i = 0; i < nsamp; ++i) {
                        size_t off = 3 * i;
                        dw[off] = (double)host_detweights[woff + off];
                        dw[off + 1] = (double)host_detweights[woff + off + 1];
                        dw[off + 2] = (double)host_detweights[woff + off + 2];
                    }
                    nfinished += 1;
                    is_done[d] = true;
                    #ifdef HAVE_NVTX
                    nvtxRangePop();
                    #endif
                }
            }
        }
    }

    // Free memory

    for (size_t d = 0; d < ndet; ++d) {
        CUDA_CHECK(hipEventDestroy(sevents[d]));
    }

    return;
}


void toast::pointing(
    int64_t nside, bool nest,
    toast::AlignedVector <double> const & boresight,
    toast::AlignedVector <double> const & hwpang,
    toast::AlignedVector <std::string> const & detnames,
    std::map <std::string, toast::AlignedVector <double> > const & detquat,
    std::map <std::string, double> const & detcal,
    std::map <std::string, double> const & deteps,
    std::map <std::string, toast::AlignedVector <int64_t> > & detpixels,
    std::map <std::string, toast::AlignedVector <double> > & detweights, size_t nobs) {

    size_t ndet = detnames.size();

    // Device query
    int ndevice;
    CUDA_CHECK(hipGetDeviceCount(&ndevice));

    // Choose first device for now
    int dev_id = 0;
    CUDA_CHECK(hipSetDevice(dev_id));

    // Find the number of SM's on this device
    int numSMs;
    CUDA_CHECK(hipDeviceGetAttribute(
        &numSMs, hipDeviceAttributeMultiprocessorCount, dev_id));

    // As a starting point, create one CUDA stream per detector.  Also create
    // one event per stream to indicate when the stream is done.
    hipStream_t streams[ndet];

    for (size_t d = 0; d < ndet; ++d) {
        CUDA_CHECK(hipStreamCreate(&(streams[d])));
    }

    // Copy common data to the GPU

    hpix * hp;
    CUDA_CHECK(hipHostMalloc(&hp, sizeof(hpix)));
    hpix_init(hp, nside);

    hpix * dev_hp;
    CUDA_CHECK(hipMalloc(&dev_hp, sizeof(hpix)));

    CUDA_CHECK(hipMemcpy(dev_hp, hp, sizeof(hpix),
                          hipMemcpyHostToDevice));

    // The maximum number of samples across all observations.  For real data
    // we would also compute the maximum number of detectors.  For this test, the
    // number of detectors is always the same.

    size_t nsamp = hwpang.size();

    // Allocate input buffers on host and device to be re-used

    double * host_boresight;
    CUDA_CHECK(hipHostMalloc(&host_boresight, 4 * nsamp * sizeof(double)));

    double * host_hwpang;
    CUDA_CHECK(hipHostMalloc(&host_hwpang, nsamp * sizeof(double)));

    double * host_detquat;
    CUDA_CHECK(hipHostMalloc(&host_detquat, ndet * 4 * sizeof(double)));

    double * dev_boresight;
    CUDA_CHECK(hipMalloc(&dev_boresight, 4 * nsamp * sizeof(double)));

    double * dev_hwpang;
    CUDA_CHECK(hipMalloc(&dev_hwpang, nsamp * sizeof(double)));

    double * dev_detquat;
    CUDA_CHECK(hipMalloc(&dev_detquat, ndet * 4 * sizeof(double)));

    // Allocate the output buffers for all detectors in device memory.  We
    // use floats for the Stokes weights and then convert to double before
    // returning.

    int64_t * dev_detpixels;
    CUDA_CHECK(hipMalloc(&dev_detpixels, ndet * nsamp * sizeof(int64_t)));

    float * dev_detweights;
    CUDA_CHECK(hipMalloc(&dev_detweights, ndet * 3 * nsamp * sizeof(float)));

    // Allocate pinned host memory for outputs

    int64_t * host_detpixels;
    CUDA_CHECK(hipHostMalloc(&host_detpixels, ndet * nsamp * sizeof(int64_t)));

    float * host_detweights;
    CUDA_CHECK(hipHostMalloc(&host_detweights,
                              ndet * 3 * nsamp * sizeof(float)));

    for (size_t ob = 0; ob < nobs; ++ob) {
        detector_pointing_healpix(nside, nest,
                                  boresight, hwpang,
                                  detnames, detquat,
                                  detcal, deteps, numSMs, streams, dev_hp,
                                  host_boresight, host_hwpang, host_detquat,
                                  dev_boresight, dev_hwpang, dev_detquat,
                                  host_detpixels, host_detweights,
                                  dev_detpixels, dev_detweights,
                                  detpixels, detweights);
    }

    // Free memory

    CUDA_CHECK(hipFree(dev_detpixels));
    CUDA_CHECK(hipFree(dev_detweights));

    CUDA_CHECK(hipFree(dev_boresight));
    CUDA_CHECK(hipFree(dev_hwpang));
    CUDA_CHECK(hipFree(dev_detquat));
    CUDA_CHECK(hipFree(dev_hp));

    CUDA_CHECK(hipHostFree(host_detpixels));
    CUDA_CHECK(hipHostFree(host_detweights));

    CUDA_CHECK(hipHostFree(host_boresight));
    CUDA_CHECK(hipHostFree(host_hwpang));
    CUDA_CHECK(hipHostFree(host_detquat));
    CUDA_CHECK(hipHostFree(hp));

    // Synchronize all streams and then destroy.
    for (size_t d = 0; d < ndet; ++d) {
        CUDA_CHECK(hipStreamSynchronize(streams[d]));
        CUDA_CHECK(hipStreamDestroy(streams[d]));
    }

    return;
}

//
// void toeplitz_multiply(
//         int fftlen, int nffts, int ncore, int nmiddle, int overlap,
//         fftw_plan & fplan, fftw_plan & rplan,
//         toast::AlignedVector <double> & fdata, toast::AlignedVector <double> & rdata,
//         toast::AlignedVector <double> const & filter,
//         toast::AlignedVector <double> & tod) {
//     // Note:  TOD buffer is replaced by output.
//     // We use "int" everywhere here since all the FFT math libraries use those.
//     // We would never take an FFT of 2^31 samples...
//
//     int nsamp = (int)tod.size();
//
//     // Clear the input buffer
//     std::fill(fdata.begin(), fdata.end(), 0.0);
//
//     std::vector <int> n_input(nffts);
//     std::vector <int> off_indata(nffts);
//     std::vector <int> off_infft(nffts);
//     std::vector <int> n_output(nffts);
//     std::vector <int> off_outdata(nffts);
//     std::vector <int> off_outfft(nffts);
//
//     int trank = 1;
//     #ifdef _OPENMP
//     trank = omp_get_thread_num();
//     #endif
//
//     if (nffts == 1) {
//         // one shot
//         n_input[0] = nsamp;
//         off_indata[0] = 0;
//         off_infft[0] = (fftlen - nsamp) >> 1;
//
//         n_output[0] = nsamp;
//         off_outdata[0] = 0;
//         off_outfft[0] = off_infft[0];
//
//         int bufoff = 0;
//
//         std::copy(&(tod[off_indata[0]]), &(tod[off_indata[0] + n_input[0]]),
//                   &(fdata[bufoff + off_infft[0]]));
//     } else {
//         // first fft
//         n_input[0] = fftlen - overlap;
//         if (n_input[0] > nsamp) {
//             n_input[0] = nsamp;
//         }
//         off_indata[0] = 0;
//         off_infft[0] = overlap;
//
//         n_output[0] = ncore;
//         off_outdata[0] = 0;
//         off_outfft[0] = overlap;
//
//         int bufoff = 0;
//
//         std::copy(&(tod[off_indata[0]]), &(tod[off_indata[0] + n_input[0]]),
//                   &(fdata[bufoff + off_infft[0]]));
//
//         // middle ffts
//
//         for (int k = 0; k < nmiddle; ++k) {
//             n_output[k + 1] = ncore;
//             off_outdata[k + 1] = (int)((nsamp - (nmiddle * ncore)) / 2) + k * ncore;
//             off_outfft[k + 1] = overlap;
//
//             n_input[k + 1] = nffts;
//             if (overlap > off_outdata[k + 1]) {
//                 off_indata[k + 1] = 0;
//             } else {
//                 off_indata[k + 1] = off_outdata[k + 1] - overlap;
//             }
//             off_infft[k + 1] = 0;
//
//             bufoff = (k + 1) * fftlen;
//             std::copy(
//                 &(tod[off_indata[k + 1]]),
//                 &(tod[off_indata[k + 1] + n_input[k + 1]]),
//                 &(fdata[bufoff + off_infft[k + 1]]));
//         }
//
//         // last fft
//         n_input[nffts - 1] = fftlen - overlap;
//         if (n_input[nffts - 1] > nsamp) {
//             n_input[nffts - 1] = nsamp;
//         }
//         off_indata[nffts - 1] = nsamp - n_input[nffts - 1];
//         off_infft[nffts - 1] = 0;
//
//         n_output[nffts - 1] = ncore;
//         off_outdata[nffts - 1] = nsamp - n_output[nffts - 1];
//         off_outfft[nffts - 1] = overlap;
//
//         bufoff = (nffts - 1) * fftlen;
//
//         std::copy(
//             &(tod[off_indata[nffts - 1]]),
//             &(tod[off_indata[nffts - 1] + n_input[nffts - 1]]),
//             &(fdata[bufoff + off_infft[nffts - 1]]));
//     }
//
//     // Forward FFTs
//
//     fftw_execute(fplan);
//
//     // Convolve with kernel
//
//     for (int k = 0; k < nffts; ++k) {
//         int bufoff = k * fftlen;
//         for (int i = 0; i < fftlen; ++i ) {
//             rdata[bufoff + i] *= filter[i];
//         }
//     }
//
//     // Reverse transform
//
//     fftw_execute(rplan);
//
//     // Copy back to TOD buffer
//
//     for (int k = 0; k < nffts; ++k) {
//         int bufoff = k * fftlen;
//         std::copy(
//             &(fdata[bufoff + off_outfft[k]]),
//             &(fdata[bufoff + off_outfft[k] + n_output[k]]),
//             &(tod[off_outdata[k]]));
//     }
//
//     return;
// }


__global__ void convert_local_pixels(
        int nsamp,
        int nsubmap,
        int nlocalsm,
        int64_t const * smlocal,
        int64_t * pixels
    ) {
    int64_t gsm;
    int64_t smpix;
    int64_t lsm;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nsamp;
         i += blockDim.x * gridDim.x) {
        gsm = pixels[i] / nsubmap;
        smpix = pixels[i] % nsubmap;
        lsm = smlocal[gsm];
        pixels[i] = (lsm * nsubmap) + smpix;
    }
    return;
}


__global__ void multiply_A(
        int nsamp,
        int nnz,
        double * tod,
        int64_t * pixels,
        float * weights,
        double * result
    ) {
    int poff;
    int toff;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nsamp;
         i += blockDim.x * gridDim.x) {
        poff = nnz * pixels[i];
        toff = nnz * i;
        tod[i] = 0.0;
        for (int j = 0; j < nnz; ++j) {
            tod[i] = weights[toff + j] * result[poff + j];
        }
    }
    return;
}


__global__ void multiply_AT(
        int nsamp,
        int nnz,
        double * tod,
        int64_t * pixels,
        float * weights,
        double * result
    ) {
    int poff;
    int toff;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nsamp;
         i += blockDim.x * gridDim.x) {
        poff = nnz * pixels[i];
        toff = nnz * i;
        for (int j = 0; j < nnz; ++j) {
            atomicAdd(
                &(result[poff + j]),
                weights[toff + j] * tod[i]);
        }
    }
    return;
}


void solver_lhs_obs(
        int64_t nside, bool nest,
        toast::AlignedVector <double> const & boresight,
        toast::AlignedVector <double> const & hwpang,
        toast::AlignedVector <double> const & filter,
        toast::AlignedVector <std::string> const & detnames,
        std::map <std::string, toast::AlignedVector <double> > const & detquat,
        std::map <std::string, double> const & detcal,
        std::map <std::string, double> const & deteps,
        int numSMs, hipStream_t * streams, hpix * dev_hp,
        double * host_boresight, double * host_hwpang, double * host_detquat,
        double * dev_boresight, double * dev_hwpang, double * dev_detquat,
        int64_t * host_detpixels, float * host_detweights,
        int64_t * dev_detpixels, float * dev_detweights,
        int fftlen, int nffts, int ncore, int nmiddle, int overlap,
        // fftw_plan * fplans, fftw_plan * rplans,
        // std::map <int, toast::AlignedVector <double> > & tfdata,
        // std::map <int, toast::AlignedVector <double> > & trdata,
        int64_t nsubmap, int64_t nnz, int64_t nsmlocal, int64_t * smlocal,
        double * dev_tod, double * dev_input, double * dev_output
    ) {

    size_t nsamp = (size_t)(boresight.size() / 4);

    size_t ndet = detnames.size();

    for (size_t d = 0; d < ndet; ++d) {
        std::memcpy(&(host_detquat[d * 4]), detquat.at(detnames[d]).data(),
                    4 * sizeof(double));
    }

    std::memcpy(host_boresight, boresight.data(), 4 * nsamp * sizeof(double));

    std::memcpy(host_hwpang, hwpang.data(), nsamp * sizeof(double));

    qa_normalize_inplace(nsamp, host_boresight);

    CUDA_CHECK(hipMemcpy(dev_boresight, host_boresight,
                          4 * nsamp * sizeof(double),
                          hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemcpy(dev_hwpang, host_hwpang, nsamp * sizeof(double),
               hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemcpy(dev_detquat, host_detquat, ndet * 4 * sizeof(double),
               hipMemcpyHostToDevice));

    // As a starting point, assume we run on the whole timestream for each
    // detector.  This may begin to approach the memory limits on the GPU,
    // at which point it is easy to add an outer loop here over chunks of
    // samples.

    // Threads per block
    int tpb = 256;

    // Blocks per Grid
    // int bpg = (int)((nsamp + tpb - 1) / tpb);
    int bpg = 32 * numSMs;

    for (size_t d = 0; d < ndet; ++d) {
        if (nest) {
            single_detector_nest <<<bpg, tpb, 0, streams[d]>>> (
                dev_hp,
                detcal.at(detnames[d]),
                deteps.at(detnames[d]),
                &(dev_detquat[4 * d]),
                nsamp,
                dev_hwpang,
                dev_boresight,
                &(dev_detpixels[d * nsamp]),
                &(dev_detweights[d * 3 * nsamp])
            );
        } else {
            single_detector_ring <<<bpg, tpb, 0, streams[d]>>> (
                dev_hp,
                detcal.at(detnames[d]),
                deteps.at(detnames[d]),
                &(dev_detquat[4 * d]),
                nsamp,
                dev_hwpang,
                dev_boresight,
                &(dev_detpixels[d * nsamp]),
                &(dev_detweights[d * nnz * nsamp])
            );
        }

        // Convert global pixels to local pixels

        convert_local_pixels <<<bpg, tpb, 0, streams[d]>>> (
            nsamp, nsubmap, nsmlocal, smlocal, &(dev_detpixels[d * nsamp])
        );

        // Sample from starting map to create timestream

        multiply_A <<<bpg, tpb, 0, streams[d]>>> (
            nsamp, nnz, &(dev_tod[d * nsamp]), &(dev_detpixels[d * nsamp]),
            &(dev_detweights[d * nnz * nsamp]), dev_input);

        // Apply Toeplitz noise covariance to TOD.




        // Accumulate to result

        multiply_A <<<bpg, tpb, 0, streams[d]>>> (
            nsamp, nnz, &(dev_tod[d * nsamp]), &(dev_detpixels[d * nsamp]),
            &(dev_detweights[d * nnz * nsamp]), dev_output);

    }

    return;
}


void toast::solver_lhs(
    int64_t nside, bool nest,
    toast::AlignedVector <double> const & boresight,
    toast::AlignedVector <double> const & hwpang,
    toast::AlignedVector <double> const & filter,
    toast::AlignedVector <std::string> const & detnames,
    std::map <std::string, toast::AlignedVector <double> > const & detquat,
    std::map <std::string, double> const & detcal,
    std::map <std::string, double> const & deteps, size_t nobs,
    toast::AlignedVector <double> & result) {

    size_t nsamp = (size_t)(boresight.size() / 4);

    size_t ndet = detnames.size();

    // Device query
    int ndevice;
    CUDA_CHECK(hipGetDeviceCount(&ndevice));

    // Choose first device for now
    int dev_id = 0;
    CUDA_CHECK(hipSetDevice(dev_id));

    // Find the number of SM's on this device
    int numSMs;
    CUDA_CHECK(hipDeviceGetAttribute(
        &numSMs, hipDeviceAttributeMultiprocessorCount, dev_id));

    // As a starting point, create one CUDA stream per detector.  Also create
    // one event per stream to indicate when the stream is done.
    hipStream_t streams[ndet];

    for (size_t d = 0; d < ndet; ++d) {
        CUDA_CHECK(hipStreamCreate(&(streams[d])));
    }

    // Copy common data to the GPU

    hpix * hp;
    CUDA_CHECK(hipHostMalloc(&hp, sizeof(hpix)));
    hpix_init(hp, nside);

    hpix * dev_hp;
    CUDA_CHECK(hipMalloc(&dev_hp, sizeof(hpix)));

    CUDA_CHECK(hipMemcpy(dev_hp, hp, sizeof(hpix),
                          hipMemcpyHostToDevice));

    // The maximum number of samples across all observations.  For real data
    // we would also compute the maximum number of detectors.  For this test, the
    // number of detectors is always the same.

    // Allocate input buffers on host and device to be re-used

    double * host_boresight;
    CUDA_CHECK(hipHostMalloc(&host_boresight, 4 * nsamp * sizeof(double)));

    double * host_hwpang;
    CUDA_CHECK(hipHostMalloc(&host_hwpang, nsamp * sizeof(double)));

    double * host_detquat;
    CUDA_CHECK(hipHostMalloc(&host_detquat, ndet * 4 * sizeof(double)));

    double * dev_boresight;
    CUDA_CHECK(hipMalloc(&dev_boresight, 4 * nsamp * sizeof(double)));

    double * dev_hwpang;
    CUDA_CHECK(hipMalloc(&dev_hwpang, nsamp * sizeof(double)));

    double * dev_detquat;
    CUDA_CHECK(hipMalloc(&dev_detquat, ndet * 4 * sizeof(double)));

    // Allocate the output buffers for all detectors in device memory.  We
    // use floats for the Stokes weights and then convert to double before
    // returning.

    int64_t * dev_detpixels;
    CUDA_CHECK(hipMalloc(&dev_detpixels, ndet * nsamp * sizeof(int64_t)));

    float * dev_detweights;
    CUDA_CHECK(hipMalloc(&dev_detweights, ndet * 3 * nsamp * sizeof(float)));

    // Allocate pinned host memory for outputs

    int64_t * host_detpixels;
    CUDA_CHECK(hipHostMalloc(&host_detpixels, ndet * nsamp * sizeof(int64_t)));

    float * host_detweights;
    CUDA_CHECK(hipHostMalloc(&host_detweights,
                              ndet * 3 * nsamp * sizeof(float)));

    // First we must pass through the pointing once in order to build up the locally
    // hit pixels.

    // Use a typical NSIDE=16 value
    int64_t nsubmap = 12 * 16 * 16;

    // We have Stokes I/Q/U values.
    int64_t nnz = 3;

    std::set <int64_t> submaps;
    submaps.clear();

    std::map <std::string, toast::AlignedVector <int64_t> > detpixels;
    std::map <std::string, toast::AlignedVector <double> > detweights;

    for (auto const & dname : detnames) {
        detpixels[dname].clear();
        detweights[dname].clear();
        detpixels[dname].resize(hwpang.size());
        detweights[dname].resize(3 * hwpang.size());
    }

    for (size_t ob = 0; ob < nobs; ++ob) {
        std::cerr << "Compute locally hit pixels:  start observation " << ob << std::endl;
        detector_pointing_healpix(nside, nest,
                                  boresight, hwpang,
                                  detnames, detquat,
                                  detcal, deteps, numSMs, streams, dev_hp,
                                  host_boresight, host_hwpang, host_detquat,
                                  dev_boresight, dev_hwpang, dev_detquat,
                                  host_detpixels, host_detweights,
                                  dev_detpixels, dev_detweights,
                                  detpixels, detweights);
        for (auto const & dname : detnames) {
            toast::update_submaps(nsubmap, detpixels[dname], submaps);
        }
        std::cerr << "Compute locally hit pixels:  stop observation " << ob << std::endl;
    }

    // Free host memory that we no longer need
    detpixels.clear();
    detweights.clear();
    CUDA_CHECK(hipHostFree(host_detpixels));
    CUDA_CHECK(hipHostFree(host_detweights));

    // Now allocate the result map
    int64_t nsmlocal = submaps.size();
    std::vector <int64_t> smlocal(nsmlocal);
    std::fill(smlocal.begin(), smlocal.end(), -1);

    int64_t sm = 0;
    for (auto const & smap : submaps) {
        smlocal[smap] = sm;
        sm++;
    }
    result.resize(nnz * nsubmap * nsmlocal);
    std::fill(result.begin(), result.end(), 1.0);

    int64_t * dev_smlocal;
    CUDA_CHECK(hipMalloc(&dev_smlocal, nsmlocal * sizeof(int64_t)));
    CUDA_CHECK(hipMemcpy(dev_smlocal, smlocal.data(), nsmlocal * sizeof(int64_t),
                          hipMemcpyHostToDevice));

    double * dev_input;
    CUDA_CHECK(hipMalloc(&dev_input, nnz * nsubmap * nsmlocal * sizeof(double)));
    CUDA_CHECK(hipMemcpy(dev_input, result.data(),
                          nnz * nsubmap * nsmlocal * sizeof(double),
                          hipMemcpyHostToDevice));

    double * dev_output;
    CUDA_CHECK(hipMalloc(&dev_output, nnz * nsubmap * nsmlocal * sizeof(double)));

    double * dev_tod;
    CUDA_CHECK(hipMalloc(&dev_tod, nsamp * sizeof(double)));

    // NOTE:  we are "cheating" here since we know that observations are all the same
    // length.  Normally we would cache all the plan lengths and batch sizes that are
    // used throughout the code...

    int npsd = filter.size();
    int half = npsd - 1;
    int fftlen = 4 * half;

    int overlap = half;

    int nffts;
    int nmiddle = 0;
    int ncore = fftlen - 2 * overlap;

    if (nsamp < (size_t)ncore) {
        // do it in one shot
        nffts = 1;
    } else {
        // we have at least the first and last ffts
        nffts = 2;

        if (nsamp <= 2 * (size_t)ncore) {
            nmiddle = 0;
        } else {
            nmiddle = (size_t)((nsamp - 2 * (size_t)ncore) / (size_t)ncore) + 1;
        }

        nffts += nmiddle;
    }

    // Compute the full symmetric fourier space kernel

    toast::AlignedVector <double> filtkern(fftlen);

    double orig_norm = 0.0;
    for (auto const & val : filter) {
        orig_norm += val;
    }

    filtkern[0] = filter[0];
    filtkern[half * 2] = filter[half];

    double norm = filtkern[0] + filtkern[half * 2];

    for (size_t i = 1; i < 2 * half; ++i) {
        size_t lower = (size_t)(i / 2);
        if (i % 2 == 0) {
            filtkern[i] = filter[lower];
        } else {
            filtkern[i] = 0.5 * (filter[lower] + filter[lower + 1]);
        }
        norm += filtkern[i];
    }

    double scale = orig_norm / norm;
    for (size_t i = 0; i <= 2 * half; ++i) {
        filtkern[i] *= scale;
    }

    // Create the FFT plans



    // Submit the solver kernels

    for (size_t ob = 0; ob < nobs; ++ob) {
        std::cerr << "Compute solver LHS:  start observation " << ob << std::endl;
        solver_lhs_obs(
            nside, nest, boresight, hwpang, filtkern, detnames,
            detquat, detcal, deteps, numSMs, streams, dev_hp, host_boresight,
            host_hwpang, host_detquat, dev_boresight, dev_hwpang, dev_detquat,
            host_detpixels, host_detweights, dev_detpixels, dev_detweights,
            fftlen, nffts, ncore, nmiddle, overlap,
            // FFT info here...
            nsubmap, nnz, nsmlocal, dev_smlocal, dev_tod, dev_input, dev_output
        );
        std::cerr << "Compute solver LHS:  stop observation " << ob << std::endl;
    }

    // Copy result from device memory back to host.


    // Synchronize streams.

    for (size_t d = 0; d < ndet; ++d) {
        CUDA_CHECK(hipStreamSynchronize(streams[d]));
    }

    // Free memory

    CUDA_CHECK(hipFree(dev_smlocal));
    CUDA_CHECK(hipFree(dev_tod));
    CUDA_CHECK(hipFree(dev_input));
    CUDA_CHECK(hipFree(dev_output));

    CUDA_CHECK(hipFree(dev_detpixels));
    CUDA_CHECK(hipFree(dev_detweights));

    CUDA_CHECK(hipFree(dev_boresight));
    CUDA_CHECK(hipFree(dev_hwpang));
    CUDA_CHECK(hipFree(dev_detquat));
    CUDA_CHECK(hipFree(dev_hp));

    CUDA_CHECK(hipHostFree(host_boresight));
    CUDA_CHECK(hipHostFree(host_hwpang));
    CUDA_CHECK(hipHostFree(host_detquat));
    CUDA_CHECK(hipHostFree(hp));

    // Destroy all streams.
    for (size_t d = 0; d < ndet; ++d) {
        CUDA_CHECK(hipStreamDestroy(streams[d]));
    }












    return;
}
